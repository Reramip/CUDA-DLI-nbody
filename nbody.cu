#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */
__global__
void bodyForce(Body *p, float dt, int n) {
  int index=threadIdx.x+blockDim.x*blockIdx.x;
  int stride=gridDim.x*blockDim.x;
  float Fx, Fy, Fz, dx, dy, dz, distSqr, invDist, invDist3;
  int i, j;
  for (i = index; i < n; i+=stride) {
     Fx = 0.0f;  Fy = 0.0f;  Fz = 0.0f;

    for (j = 0; j < n; j++) {
       dx = p[j].x - p[i].x;
       dy = p[j].y - p[i].y;
       dz = p[j].z - p[i].z;
       distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
       invDist = rsqrtf(distSqr);
       invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }
    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

__global__
void integrate(Body *p, float dt, int n) { // integrate position
  int index=threadIdx.x+blockDim.x*blockIdx.x;
  int stride=gridDim.x*blockDim.x;
  for (int i = index ; i < n; i+=stride) { 
    p[i].x += p[i].vx*dt;
    p[i].y += p[i].vy*dt;
    p[i].z += p[i].vz*dt;
  }
}

int main(const int argc, const char** argv) {

  /*
   * Do not change the value for `nBodies` here. If you would like to modify it,
   * pass values into the command line.
   */

  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

  if (argc > 2) salt = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;

  //buf = (float *)malloc(bytes);
  hipMallocManaged(&buf, bytes);

  Body *p = (Body*)buf;

  int deviceId;
  int smNum;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&smNum,hipDeviceAttributeMultiprocessorCount,deviceId);

  size_t threadNum=256;
  size_t blockNum=32*smNum;

  /*
   * As a constraint of this exercise, `randomizeBodies` must remain a host function.
   */

  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

  double totalTime = 0;

  hipMemPrefetchAsync(buf, bytes, deviceId);

  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

  /*******************************************************************/
  // Do not modify these 2 lines of code.
  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();
  /*******************************************************************/

    //hipStream_t stream;
    //hipStreamCreate(&stream);

  /*
   * You will likely wish to refactor the work being done in `bodyForce`,
   * as well as the work to integrate the positions.
   */
    bodyForce<<<blockNum, threadNum>>>(p, dt, nBodies); // compute interbody forces
    //hipDeviceSynchronize();
  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */
    integrate<<<blockNum, threadNum>>>(p, dt, nBodies); // integrate position
    //hipDeviceSynchronize();
    // parallelize this for
    /*for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }*/
    hipDeviceSynchronize();
    
    //hipStreamDestroy(stream);

  /*******************************************************************/
  // Do not modify the code in this section.
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
  checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
  checkAccuracy(buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/

  /*
   * Feel free to modify code below.
   */

  hipFree(buf);
}
